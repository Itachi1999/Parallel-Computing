// Launch config for huge data
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define BLOCKSIZE 1024

__global__ void dkernel(unsigned *vector, unsigned vectorSize)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < vectorSize) vector[id] = id;
}

int main(int nn, char *str[])
{
    unsigned N = atoi(str[1]);
    unsigned *vector, *hvector;
    unsigned i = 0;

    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned *)malloc(N * sizeof(unsigned));

    unsigned nBlocks = ceil((float)N / BLOCKSIZE);
    printf("nBlocks = %d\n", nBlocks);

    dkernel<<<nBlocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++)
    {
        printf("%d ", hvector[i]);
    }

    return 0;
}