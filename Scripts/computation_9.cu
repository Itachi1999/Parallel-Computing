// 1D version of the code

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define N 5
#define M 6

__global__ void dkernel(unsigned *mat)
{
    unsigned xId = blockIdx.x, yId = threadIdx.x, col = blockDim.x;

    mat[xId * col + yId] = xId * col + yId;
}

int main()
{
    //dim3 block(N, M, 1);
    unsigned *matrix, *hmatrix, i = 0, j = 0;

    hipMalloc(&matrix, N * M * sizeof(unsigned));
    hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));

    dkernel<<<N, M>>>(matrix);
    hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++)
    {
        for(j = 0; j < M; j++)
        {
            printf("%d\t", hmatrix[i * M + j]);
        }
        printf("\n");
    }
    
    return 0;
}