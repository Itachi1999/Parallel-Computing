#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 100

__global__ void func(int *a)
{
    a[threadIdx.x] = threadIdx.x * threadIdx.x;
}


//This won't work since GPU and CPU will have different memory and the array is assigned in CPU
//The GPU can't access the same memory

// int main()
// {
//     int a[N] = {0}, i = 0;
    
//     func<<<1, N>>>(a);

//     cudaDeviceSynchronize();
//     for(i = 0; i < N; i++)
//     {
//         printf("%d ", a[i]);
//     }

//     return 0;
// }


int main()
{
    int a[N] = {0}, *da, i = 0;
    
    hipMalloc(&da, sizeof(int) * N);
    func<<<1, N>>>(da);
    hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost);

    //cudaDeviceSyncronize() not needed since cudaMemory halts the CPU and then goes to the queue of GPU

    for(i = 0; i < N; i++)
    {
       printf("%d ", a[i]);
    }

    return 0;
}