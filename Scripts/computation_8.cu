// 2D version of the code

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define N 5
#define M 6

__global__ void dkernel(unsigned *mat)
{
    unsigned xId = threadIdx.x, yId = threadIdx.y;

    mat[xId * M + yId] = xId * M + yId;
}

int main()
{
    dim3 block(N, M, 1);
    unsigned *matrix, *hmatrix, i = 0, j = 0;

    hipMalloc(&matrix, N * M * sizeof(unsigned));
    hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));

    dkernel<<<1, block>>>(matrix);
    hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++)
    {
        for(j = 0; j < M; j++)
        {
            printf("%d\t", hmatrix[i * M + j]);
        }
        printf("\n");
    }
    
    return 0;
}