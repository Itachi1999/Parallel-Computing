#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100

//Sequential

// __global__ void dkernel()
// {
//     int i = 0;
//     for(i = 0; i < N; i++)
//     {
//         printf("%d\n", i * i);    
//     }
// }

// int main()
// {
//     dkernel<<<1, 1>>>();
//     cudaDeviceSynchronize();
//     return 0;
// }


//Parallel


//Thread Ids always start from zero and get increased sequentially
__global__ void func()
{
    printf("%d\n", threadIdx.x * threadIdx.x);
}

int main()
{
    func<<<1, N>>>();
    hipDeviceSynchronize();

    return 0;
}

