#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel()
{
    printf("Hello World! \n");
}

int main()
{
    dkernel<<<1, 32>>>(); //32 threads within 1 thread block
    hipDeviceSynchronize();

    return 0;
}